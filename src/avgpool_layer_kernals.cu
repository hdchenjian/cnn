#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "avgpool_layer.h"
#include "hip/hip_runtime.h"
}

__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int k = id % c;
    id /= c;
    int b = id;

    int i;
    int out_index = (k + c*b);
    output[out_index] = 0;
    for(i = 0; i < w*h; ++i){
        int in_index = i + h*w*(k + b*c);
        output[out_index] += input[in_index];
    }
    output[out_index] /= w*h;
}

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int k = id % c;
    id /= c;
    int b = id;

    int i;
    int out_index = (k + c*b);
    for(i = 0; i < w*h; ++i){
        int in_index = i + h*w*(k + b*c);
        in_delta[in_index] += out_delta[out_index] / (w*h);
    }
}

extern "C" void forward_avgpool_layer_gpu(const avgpool_layer *l, float *input)
{
    size_t n = l->c*l->batch;
    forward_avgpool_layer_kernel<<<cuda_gridsize(n), BLOCK>>>(n, l->w, l->h, l->c, input, l->output_gpu);
    check_error(hipPeekAtLastError());
}

extern "C" void backward_avgpool_layer_gpu(const avgpool_layer *l, float *delta)
{
    size_t n = l->c*l->batch;
    backward_avgpool_layer_kernel<<<cuda_gridsize(n), BLOCK>>>(n, l->w, l->h, l->c, delta, l->delta_gpu);
    check_error(hipPeekAtLastError());
}

